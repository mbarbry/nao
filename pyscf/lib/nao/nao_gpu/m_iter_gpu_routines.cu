#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include<sys/param.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "m_iter_gpu_routines.h"

float *X4_d, *ksn2e_d, *ksn2f_d;
float *v_ext_d;
float *vdp_d, *sab_d, *nb2v_d;
int norbs, nfermi, vstart, nprod;
scsr_matrix cc_da_d, v_dab_d;
hipsparseHandle_t handle_cuparse=0;
hipblasHandle_t handle_cublas;


int sum_int_vec(int *mat, int N)
{
  int i;
  int sum_int = 0;

  for (i=0; i<N; i++)
  { 
    sum_int += mat[i];
  }

  return sum_int;
}

float sum_float_vec(float *mat, int N)
{
  int i;
  float sum_float = 0;

  for (i=0; i<N; i++)
  { 
    sum_float += mat[i];
  }

  return sum_float;
}


/*
  initialize sparse matrix on the gpu
*/
extern "C" scsr_matrix init_sparse_matrix_csr_gpu_float(float *csrValA, int *csrRowPtrA, 
    int *csrColIndA, int m, int n, int nnz, int RowPtrSize)
{

  scsr_matrix csr;

  csr.m = m;
  csr.n = n;
  csr.nnz = nnz;
  csr.RowPtrSize = RowPtrSize;

  checkCudaErrors(hipMalloc( (void **)&csr.data, sizeof(float) * nnz));
  checkCudaErrors(hipMalloc( (void **)&csr.ColInd, sizeof(float) * nnz));
  checkCudaErrors(hipMalloc( (void **)&csr.RowPtr, sizeof(int) * RowPtrSize));

  checkCudaErrors(hipMemcpy( csr.data, csrValA, sizeof(float) * nnz, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy( csr.ColInd, csrColIndA, sizeof(int) * nnz, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy( csr.RowPtr, csrRowPtrA, sizeof(int) * RowPtrSize, hipMemcpyHostToDevice));

  // Initialize and setup matrix descriptor
  checkCudaErrors(hipsparseCreateMatDescr(&csr.descr)); 
  
  checkCudaErrors(hipsparseSetMatType(csr.descr,HIPSPARSE_MATRIX_TYPE_GENERAL));
  checkCudaErrors(hipsparseSetMatIndexBase(csr.descr, HIPSPARSE_INDEX_BASE_ZERO));  

  return csr;
}

extern "C" void free_csr_matrix_gpu(scsr_matrix csr)
{
  checkCudaErrors(hipFree(csr.data));
  checkCudaErrors(hipFree(csr.ColInd));
  checkCudaErrors(hipFree(csr.RowPtr));

  checkCudaErrors(hipsparseDestroyMatDescr(csr.descr));
}

extern "C" void init_tddft_iter_gpu(float *X4, int norbs_in, float *ksn2e,
                  float *ksn2f, int nfermi_in, int nprod_in, int vstart_in,
                  float *cc_da_vals, int *cc_da_rowPtr, int *cc_da_col_ind,
                  int *cc_da_shape, int cc_da_nnz, int cc_da_indptr_size,
                  float *v_dab_vals, int *v_dab_rowPtr, int *v_dab_col_ind,
                  int *v_dab_shape, int v_dab_nnz, int v_dab_indptr_size)
{

  norbs = norbs_in;
  nfermi = nfermi_in;
  nprod = nprod_in;
  vstart = vstart_in;

  printf("v_dab_indptr_size = %d, norbs=%d\n", v_dab_indptr_size, norbs);
  // init sparse matrices on GPU
  cc_da_d = init_sparse_matrix_csr_gpu_float(cc_da_vals, cc_da_rowPtr, 
                  cc_da_col_ind, cc_da_shape[0], cc_da_shape[1], cc_da_nnz, cc_da_indptr_size);

  v_dab_d = init_sparse_matrix_csr_gpu_float(v_dab_vals, v_dab_rowPtr, 
                  v_dab_col_ind, v_dab_shape[0], v_dab_shape[1], v_dab_nnz, v_dab_indptr_size);
  printf("v_dab.shape = %d, %d\n", v_dab_d.m, v_dab_d.n);

  checkCudaErrors(hipsparseCreate(&handle_cuparse));
  checkCudaErrors(hipblasCreate(&handle_cublas));

  checkCudaErrors(hipMalloc( (void **)&X4_d, sizeof(float) * norbs*norbs));
  checkCudaErrors(hipMalloc( (void **)&ksn2e_d, sizeof(float) * norbs));
  checkCudaErrors(hipMalloc( (void **)&ksn2f_d, sizeof(float) * norbs));
  
  checkCudaErrors(hipMalloc( (void **)&v_ext_d, sizeof(float) * nprod));
  checkCudaErrors(hipMalloc( (void **)&vdp_d, sizeof(float) * cc_da_d.m));
  checkCudaErrors(hipMalloc( (void **)&sab_d, sizeof(float) * v_dab_d.n));
  checkCudaErrors(hipMalloc( (void **)&nb2v_d, sizeof(float) * nfermi*norbs));

  checkCudaErrors(hipMemcpy( X4_d, X4, sizeof(float) * norbs*norbs, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy( ksn2e_d, ksn2e, sizeof(float) * norbs, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy( ksn2f_d, ksn2f, sizeof(float) * norbs, hipMemcpyHostToDevice));

}

extern "C" void free_device()
{

  checkCudaErrors(hipFree(X4_d));
  checkCudaErrors(hipFree(ksn2e_d));
  checkCudaErrors(hipFree(ksn2f_d));

  checkCudaErrors(hipFree(v_ext_d));
  checkCudaErrors(hipFree(vdp_d));
  checkCudaErrors(hipFree(sab_d));
  checkCudaErrors(hipFree(nb2v_d));

  free_csr_matrix_gpu(cc_da_d);
  free_csr_matrix_gpu(v_dab_d);

  checkCudaErrors(hipsparseDestroy(handle_cuparse));
  checkCudaErrors(hipblasDestroy(handle_cublas));
}

extern "C" void apply_rf0_device(float *v_ext_real, float *v_ext_imag, float *temp)
{
  float alpha = 1.0, beta = 0.0;

  // real part first


  checkCudaErrors(hipMemcpy( v_ext_d, v_ext_real, sizeof(float) * nprod, hipMemcpyHostToDevice));

  /*
     hipsparseScsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, 
        int m, int n, int nnz, const float           *alpha, 
        const hipsparseMatDescr_t descrA, 
        const float           *csrValA, 
        const int *csrRowPtrA, const int *csrColIndA,
        const float           *x, const float           *beta, 
        float           *y)
  */
  checkCudaErrors(hipsparseScsrmv(handle_cuparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        cc_da_d.m, cc_da_d.n, cc_da_d.nnz, &alpha,
        cc_da_d.descr, cc_da_d.data, cc_da_d.RowPtr, cc_da_d.ColInd, 
        v_ext_d, &beta, vdp_d));

  checkCudaErrors(hipsparseScsrmv(handle_cuparse, HIPSPARSE_OPERATION_TRANSPOSE,
        v_dab_d.m, v_dab_d.n, v_dab_d.nnz, &alpha,
        v_dab_d.descr, v_dab_d.data, v_dab_d.RowPtr, v_dab_d.ColInd, 
        vdp_d, &beta, sab_d));

  float *X4, *xocc, *xocc_d;
  X4 = (float*) malloc(sizeof(float)*norbs*norbs);
  xocc = (float*) malloc(sizeof(float)*nfermi*norbs);

  checkCudaErrors(hipMemcpy( X4, X4_d, sizeof(float) * norbs*norbs, hipMemcpyDeviceToHost));
  int i, j;
  printf("xocc = \n");
  for (i=0; i<nfermi; i++)
  {
    for (j=0; j< norbs; j++)
    {
      //printf("  %f", X4[i*norbs + j]);
      xocc[i*norbs + j] = X4[i*norbs + j];
    }
    printf("\n");
  }
  free(X4);
  
  checkCudaErrors(hipMalloc( (void **)&xocc_d, sizeof(float) * nfermi*norbs));
  checkCudaErrors(hipMemcpy( xocc_d, xocc, sizeof(float) * nfermi*norbs, hipMemcpyHostToDevice));

  free(xocc);


  checkCudaErrors(hipblasSgemm(handle_cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, nfermi, norbs, norbs, &alpha, xocc_d, nfermi,
        sab_d, norbs, &beta, nb2v_d, nfermi));
  //checkCudaErrors(hipblasSgemm(handle_cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, nvirt, nocc, norb, &alpha, &aux_X4_mat_d[(Fmin-1)*norb], norb, XVV_mat_d,
  //     norb, &beta, XXVV_mat_im_d, nvirt));

  checkCudaErrors(hipMemcpy( temp, nb2v_d, sizeof(float) * nfermi*norbs, hipMemcpyDeviceToHost));
  
  checkCudaErrors(hipFree(xocc_d));

  /*
  int *RowPtr, *ColInd;
  float *data;

  RowPtr = (int *) malloc(sizeof(int)*cc_da_d.RowPtrSize);
  ColInd = (int *) malloc(sizeof(int)*cc_da_d.nnz);
  data = (float *) malloc(sizeof(float)*cc_da_d.nnz);
  
  checkCudaErrors(hipMemcpy( RowPtr, cc_da_d.RowPtr, sizeof(int) * cc_da_d.RowPtrSize, hipMemcpyDeviceToHost));
  int sum_rowPtr = sum_int_vec(RowPtr, cc_da_d.RowPtrSize);
  
  checkCudaErrors(hipMemcpy( ColInd, cc_da_d.ColInd, sizeof(int) * cc_da_d.nnz, hipMemcpyDeviceToHost));
  int sum_colInd = sum_int_vec(ColInd, cc_da_d.nnz);
  
  checkCudaErrors(hipMemcpy( data, cc_da_d.data, sizeof(float) * cc_da_d.nnz, hipMemcpyDeviceToHost));
  float sum_data = sum_float_vec(data, cc_da_d.nnz);
*/

  /*
  printf("cc_da : gpu\n");
  printf("m = %d, n = %d, nnz = %d\n", cc_da_d.m, cc_da_d.n, cc_da_d.nnz);
  printf("sum_ind : %d, %d\n", sum_rowPtr, sum_colInd);
  printf("sum data: %f\n", sum_data);

  free(data);
  free(RowPtr);
  free(ColInd);
  */

}
